#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <unordered_map>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <algorithm>
#include <iostream>
#include <direct.h>
#include "newsketch.cuh"
#include "fstream"
#include "GeneralUtil.cuh"
//#include "sm_35_atomic_functions.h"
#include<vector>
typedef unsigned long long uint64;
typedef unsigned int uint32;
typedef unsigned char uint8;

using namespace std;
GeneralSketchBloom *GSB;
GeneralSketchBloom *GSB1;
GeneralVSketch *GVS;
GeneralVSketch *GVS1;
unsigned int *packetdata;
__global__ void initsketch(GeneralSketchBloom *GSB,GeneralVSketch *GVS, int GSB_or_GVS,int size_or_spread,int number,unsigned int* packetdata) {
	int id = blockIdx.x *blockDim.x + threadIdx.x;
	if (id >= number) return;
	uint32_t srcIP =  packetdata[id * 2];
	uint32_t dstIP =  packetdata[id * 2 + 1];
	uint32_t x;
	uint32_t j;
	if (GSB_or_GVS == 0) {
		for (int i = 0; i < 1; i++) {
			x = uIntHash(srcIP);
			if (size_or_spread == 0) {
				if (*GSB->sketchName == 0) {
					for (int pp = 0; pp < 4; pp++) {
						j = (intHash(x^GSB->S[pp]) % *GSB->w + *GSB->w) % *GSB->w;
						encodeCounter(GSB->C[0][j]);
					}
				}
				else
					if (*GSB->sketchName == 1) {

						for (int pp = 0; pp < 4; pp++) {
							j = (intHash(x ^ GSB->S[pp]) % *GSB->w + *GSB->w) % *GSB->w;
							encodeBitmap(GSB->B[0][j]);
						}
					}
					else
						if (*GSB->sketchName == 2) {
							for (int pp = 0; pp < 4; pp++) {
								j = (intHash(x^GSB->S[pp]) % *GSB->w + *GSB->w) % *GSB->w;
								encodeFMsketch(GSB->F[0][j]);
							}
						}
						else
							if (*GSB->sketchName == 3) {
								for (int pp = 0; pp < 4; pp++) {
									j = (intHash(x^GSB->S[pp]) % *GSB->w + *GSB->w) % *GSB->w;
									encodeHyperLogLog(GSB->H[0][j]);
								}
							}
			}
			else {
				if (*GSB->sketchName == 0) {
					for (int pp = 0; pp < 4; pp++) {
						j = (intHash(x^GSB->S[pp]) % *GSB->w + *GSB->w) % *GSB->w;
						encodeCounterEID(GSB->C[0][j], srcIP);
					}
				}
				else
					if (*GSB->sketchName == 1) {
						for (int pp = 0; pp < 4; pp++) {
							j = (intHash(x^GSB->S[pp]) % *GSB->w + *GSB->w) % *GSB->w;
							encodeBitmapEID(GSB->B[0][j], srcIP);
						}
					}
					else
						if (*GSB->sketchName == 2) {
							for (int pp = 0; pp < 4; pp++) {
								j = (intHash(x^GSB->S[pp]) % *GSB->w + *GSB->w) % *GSB->w;
								encodeFMsketchEID(GSB->F[0][j], srcIP);
							}
						}
						else
							if (*GSB->sketchName == 3) {
								for (int pp = 0; pp < 4; pp++) {
									j = (intHash(x^GSB->S[pp]) % *GSB->w + *GSB->w) % *GSB->w;
									encodeHyperLogLogEID(GSB->H[0][j], srcIP);
								}
							}
			}
		}
	}
	else if(GSB_or_GVS==1){
		int w_m = *GVS->w / *GVS->m;
		for (int i = 0; i < 1; i++) {
			if (size_or_spread == 0) {
				if (*GVS->sketchName == 0)
					encodeCounterSegment(GVS->C[0], srcIP, GVS->S, w_m);
				else
					if (*GVS->sketchName == 1)
						encodeBitmapSegment(GVS->B[0], srcIP, GVS->S, w_m);
					else
						if (*GVS->sketchName == 2)
							encodeFMsketchSegment(GVS->F[0], srcIP, GVS->S, w_m);
						else
							if (*GVS->sketchName == 3)
								encodeHyperLogLogSegment(GVS->H[0], srcIP, GVS->S, w_m);
			}
			else {
				if (*GVS->sketchName == 0)
					encodeCounterSegmentEID(GVS->C[0], srcIP, dstIP, GVS->S, w_m);
				else
					if (*GVS->sketchName == 1)
						encodeBitmapSegmentEID(GVS->B[0], srcIP, dstIP, GVS->S, w_m);
					else
						if (*GVS->sketchName == 2)
							encodeFMsketchSegmentEID(GVS->F[0], srcIP, dstIP, GVS->S, w_m);
						else
							if (*GVS->sketchName == 3)
								encodeHyperLogLogSegmentEID(GVS->H[0], srcIP, dstIP, GVS->S, w_m);
			}
		}
	}
}

void getoutputGSB(GeneralSketchBloom *GSB,int sketch_name,int size_or_spread) {
	int w;
	GeneralSketchBloom tmp1;
	hipMemcpy(&tmp1, GSB, sizeof(GeneralSketchBloom), hipMemcpyDeviceToHost);
	hipMemcpy(&w, tmp1.w, sizeof(int), hipMemcpyDeviceToHost);
	if (sketch_name == 0) {
		Counter ***x = (Counter ***)malloc(sizeof(Counter **) * 1);
		hipMemcpy(x, tmp1.C, sizeof(Counter **), hipMemcpyDeviceToHost);
		Counter **tmpC = (Counter **)malloc(w * sizeof(Counter *));
		hipMemcpy(tmpC, x[0], sizeof(Counter *)* w, hipMemcpyDeviceToHost);
		int GSBS[4];
		hipMemcpy(GSBS, tmp1.S, sizeof(int) * 4, hipMemcpyDeviceToHost);
		ofstream out;
		if (size_or_spread == 0)
			out.open("..\\..\\result\\BSketch\\gpu_counter_size_out.txt", ios::out);
		else
			out.open("..\\..\\result\\BSketch\\gpu_counter_spread_out.txt", ios::out);
		for (int i = 0; i < w;i++) {
			Counter p;
			hipMemcpy(&p, tmpC[i], sizeof(Counter), hipMemcpyDeviceToHost);
			int m;
			hipMemcpy(&m, p.m, sizeof(int), hipMemcpyDeviceToHost);
			//cout << m << endl;
			int *o=(int *)malloc(sizeof(int)*m);
			hipMemcpy(o, p.counters, m * sizeof(int), hipMemcpyDeviceToHost);
			for (int j = 0; j < m; j++) {
				out << o[j];
				out << endl;
			}
		}
		out.close();
	}
	else 
		if (sketch_name == 1) {
			Bitmap ***x = (Bitmap ***)malloc(sizeof(Bitmap **) * 1);
			hipMemcpy(x, tmp1.B, sizeof(Bitmap **), hipMemcpyDeviceToHost);
			Bitmap **tmpB = (Bitmap **)malloc(w * sizeof(Bitmap *));
			hipMemcpy(tmpB, x[0], sizeof(Bitmap *)* w, hipMemcpyDeviceToHost);
			int GSBS[4];
			hipMemcpy(GSBS, tmp1.S, sizeof(int) * 4, hipMemcpyDeviceToHost);
			ofstream out;
			if (size_or_spread == 0)
				out.open("..\\..\\result\\BSketch\\gpu_bitmap_size_out.txt", ios::out);
			else
				out.open("..\\..\\result\\BSketch\\gpu_bitmap_spread_out.txt", ios::out);
			for (int i = 0; i < w;i++) {
				Bitmap p;
				hipMemcpy(&p, tmpB[i], sizeof(Bitmap), hipMemcpyDeviceToHost);
				int m;
				hipMemcpy(&m, p.m, sizeof(int), hipMemcpyDeviceToHost);
				//cout << m << endl;
				bool *o=(bool *)malloc(sizeof(bool)*m);
				hipMemcpy(o, p.B, m * sizeof(bool), hipMemcpyDeviceToHost);
				for (int j = 0; j < m; j++) {
					out << o[j];
					out << endl;
				}
			}
			out.close();
		}
		else
			if (sketch_name == 2) {
				FMsketch ***x = (FMsketch ***)malloc(sizeof(FMsketch **) * 1);
				hipMemcpy(x, tmp1.F, sizeof(FMsketch **), hipMemcpyDeviceToHost);
				FMsketch **tmpF = (FMsketch **)malloc(w * sizeof(FMsketch *));
				hipMemcpy(tmpF, x[0], sizeof(FMsketch *)* w, hipMemcpyDeviceToHost);
				int GSBS[4];
				hipMemcpy(GSBS, tmp1.S, sizeof(int) * 4, hipMemcpyDeviceToHost);
				ofstream out;
				if (size_or_spread == 0)
					out.open("..\\..\\result\\BSketch\\gpu_fm_size_out.txt", ios::out);
				else
					out.open("..\\..\\result\\BSketch\\gpu_fm_spread_out.txt", ios::out);
				for (int i = 0; i < w;i++) {
					FMsketch p;
					hipMemcpy(&p, tmpF[i], sizeof(FMsketch), hipMemcpyDeviceToHost);
					int m;
					hipMemcpy(&m, p.m, sizeof(int), hipMemcpyDeviceToHost);
					int size;			
					hipMemcpy(&size, p.FMsketchSize, sizeof(int), hipMemcpyDeviceToHost);
					//cout << m << " " << size << endl;
					bool **o=(bool **)malloc(sizeof(bool *)*m);
					hipMemcpy(o, p.FMsketchMatrix, m * sizeof(bool*), hipMemcpyDeviceToHost);
					for (int j = 0; j < m; j++) {
						bool *r= (bool *)malloc(sizeof(bool)*size);
						hipMemcpy(r, o[j], size * sizeof(bool), hipMemcpyDeviceToHost);
						for (int l = 0; l < size; l++) {
							out << r[l];
							out << endl;
						}
					}
				}
				out.close();
			}
			else
				if (sketch_name == 3) {
					HyperLogLog ***x = (HyperLogLog ***)malloc(sizeof(HyperLogLog **) * 1);
					hipMemcpy(x, tmp1.H, sizeof(HyperLogLog **), hipMemcpyDeviceToHost);
					HyperLogLog **tmpH = (HyperLogLog **)malloc(w * sizeof(HyperLogLog *));
					hipMemcpy(tmpH, x[0], sizeof(HyperLogLog *)* w, hipMemcpyDeviceToHost);
					int GSBS[4];
					hipMemcpy(GSBS, tmp1.S, sizeof(int) * 4, hipMemcpyDeviceToHost);
					ofstream out;
					if (size_or_spread == 0)
						out.open("..\\..\\result\\BSketch\\gpu_hll_size_out.txt", ios::out);
					else
						out.open("..\\..\\result\\BSketch\\gpu_hll_spread_out.txt", ios::out);
					for (int i = 0; i < w; i++) {
						HyperLogLog p;
						hipMemcpy(&p, tmpH[i], sizeof(HyperLogLog), hipMemcpyDeviceToHost);
						int m;
						hipMemcpy(&m, p.m, sizeof(int), hipMemcpyDeviceToHost);
						int size;
						hipMemcpy(&size, p.HLLSize, sizeof(int), hipMemcpyDeviceToHost);
						//cout << m << " " << size << endl;
						bool **o = (bool **)malloc(sizeof(bool *)*m);
						hipMemcpy(o, p.HLL, m * sizeof(bool*), hipMemcpyDeviceToHost);
						for (int j = 0; j < m; j++) {
							bool *r = (bool *)malloc(sizeof(bool)*size);
							hipMemcpy(r, o[j], size * sizeof(bool), hipMemcpyDeviceToHost);
							for (int l = 0; l < size; l++) {
								out << r[l];
								out << endl;
							}
						}
					}
					out.close();
				}
}
void getoutputGVS(GeneralVSketch *GVS,int sketch_name,int size_or_spread) {
	int w;
	GeneralVSketch tmp1;
	hipMemcpy(&tmp1, GVS, sizeof(GeneralVSketch), hipMemcpyDeviceToHost);
	hipMemcpy(&w, tmp1.w, sizeof(int), hipMemcpyDeviceToHost);
	//cout << w << endl;
	if (sketch_name == 0) {
		Counter **x = (Counter **)malloc(sizeof(Counter *) * 1);
		hipMemcpy(x, tmp1.C, sizeof(Counter *) * 1, hipMemcpyDeviceToHost);
		ofstream out;
		if(size_or_spread==0)
			out.open("..\\..\\result\\VSketch\\gpu_counter_size_out.txt", ios::out);
		else
			out.open("..\\..\\result\\VSketch\\gpu_counter_spread_out.txt", ios::out);
		Counter p;
		hipMemcpy(&p, x[0], sizeof(Counter), hipMemcpyDeviceToHost);
		int m;
		hipMemcpy(&m, p.m, sizeof(int), hipMemcpyDeviceToHost);
		//cout << m << endl;
		int *o = (int *)malloc(sizeof(int)*m);
		hipMemcpy(o, p.counters, m * sizeof(int), hipMemcpyDeviceToHost);
		for (int j = 0; j < m; j++) {
			out << o[j];
			out << endl;
		}
		out.close();
	}
	else
		if (sketch_name == 1) {
			Bitmap **x = (Bitmap **)malloc(sizeof(Bitmap *) * 1);
			hipMemcpy(x, tmp1.B, sizeof(Bitmap *) * 1, hipMemcpyDeviceToHost);
			ofstream out;
			if (size_or_spread == 0)
				out.open("..\\..\\result\\VSketch\\gpu_bitmap_size_out.txt", ios::out);
			else
				out.open("..\\..\\result\\VSketch\\gpu_bitmap_spread_out.txt", ios::out);
			Bitmap p;
			hipMemcpy(&p, x[0], sizeof(Bitmap), hipMemcpyDeviceToHost);
			int m;
			hipMemcpy(&m, p.m, sizeof(int), hipMemcpyDeviceToHost);
			//cout << m << endl;
			bool *o = (bool *)malloc(sizeof(bool)*m);
			hipMemcpy(o, p.B, m * sizeof(bool), hipMemcpyDeviceToHost);
			for (int j = 0; j < m; j++) {
				out << o[j];
				out << endl;
			}
			out.close();
		}
		else
			if (sketch_name == 2) {
				FMsketch **x = (FMsketch **)malloc(sizeof(FMsketch *) * 1);
				hipMemcpy(x, tmp1.F, sizeof(FMsketch *), hipMemcpyDeviceToHost);
				ofstream out;
				if (size_or_spread == 0)
					out.open("..\\..\\result\\VSketch\\gpu_fm_size_out.txt", ios::out);
				else
					out.open("..\\..\\result\\VSketch\\gpu_fm_spread_out.txt", ios::out);
				FMsketch p;
				hipMemcpy(&p, x[0], sizeof(FMsketch), hipMemcpyDeviceToHost);
				int m;
				hipMemcpy(&m, p.m, sizeof(int), hipMemcpyDeviceToHost);
				int size;
				hipMemcpy(&size, p.FMsketchSize, sizeof(int), hipMemcpyDeviceToHost);
				//cout << m << " " << size << endl;
				bool **o = (bool **)malloc(sizeof(bool *)*m);
				hipMemcpy(o, p.FMsketchMatrix, m * sizeof(bool*), hipMemcpyDeviceToHost);
				for (int j = 0; j < m; j++) {
					bool *r = (bool *)malloc(sizeof(bool)*size);
					hipMemcpy(r, o[j], size * sizeof(bool), hipMemcpyDeviceToHost);
					for (int l = 0; l < size; l++) {
						out << r[l];
						out << endl;
					}
				}
				out.close();
			}
			else
				if (sketch_name == 3) {
					HyperLogLog **x = (HyperLogLog **)malloc(sizeof(HyperLogLog *) * 1);
					hipMemcpy(x, tmp1.H, sizeof(HyperLogLog **), hipMemcpyDeviceToHost);
					ofstream out;
					if (size_or_spread == 0)
						out.open("..\\..\\result\\VSketch\\gpu_hll_size_out.txt", ios::out);
					else
						out.open("..\\..\\result\\VSketch\\gpu_hll_spread_out.txt", ios::out);
					HyperLogLog p;
					hipMemcpy(&p, x[0], sizeof(HyperLogLog), hipMemcpyDeviceToHost);
					int m;
					hipMemcpy(&m, p.m, sizeof(int), hipMemcpyDeviceToHost);
					int size;
					hipMemcpy(&size, p.HLLSize, sizeof(int), hipMemcpyDeviceToHost);
					//cout << m << " " << size << endl;
					bool **o = (bool **)malloc(sizeof(bool *)*m);
					hipMemcpy(o, p.HLL, m * sizeof(bool*), hipMemcpyDeviceToHost);
					for (int j = 0; j < m; j++) {
						bool *r = (bool *)malloc(sizeof(bool)*size);
						hipMemcpy(r, o[j], size * sizeof(bool), hipMemcpyDeviceToHost);
						for (int l = 0; l < size; l++) {
							out << r[l];
							out << endl;
						}
					}
					out.close();
				}
}

int readdata(string filename,char** data) {
	ifstream in;
	in.open(filename);
	
	char t[50];
	int number=0;
	while (in.getline(t, 40)) number++;
	in.close();
	char** tmpdata;
	tmpdata = (char **)malloc(sizeof(char *)*number);
	hipMalloc((void **)&data,sizeof(char *)*number);
	in.open(filename);
	for (int i = 0; i < number;i++) {
		in.getline(t, 40);
		hipMalloc((void **)&tmpdata[i], sizeof(char) * 40);
		hipMemcpy(tmpdata[i],t,sizeof(char)*40,hipMemcpyHostToDevice);
	}
	hipMemcpy(data, tmpdata, sizeof(char *)*number, hipMemcpyHostToDevice);
	in.close();
	return number;
}

void ip_str_to_num1(unsigned int *src, unsigned int *dst, char *buf) {
	sscanf(buf, "%u%u", src,dst);
}

unsigned int *readdata1(int *number,string filename,unsigned int* data) {
	ifstream in;
	in.open(filename);
	char t[50];
	*number = 0;
	while (in.getline(t, 40)) (*number)++;
	in.close();
	cout << *number << endl;
	size_t pitch=0;
	unsigned int *tmpdata;
	hipMalloc((void **)&data, sizeof(unsigned int)*2* *number);
	tmpdata = (unsigned int *)malloc(sizeof(unsigned int) * 2 * *number);
	in.open(filename);
	unsigned int x, y;
	for (int i = 0; i < *number; i++) {
		in.getline(t, 40);
		ip_str_to_num1(&x, &y, t);
		tmpdata[i * 2] = x;
		tmpdata[i * 2 + 1] = y;
		if (i % 1000000 == 0) cout << i<<endl;
	}
	in.close();
	hipMemcpy(data, tmpdata,sizeof(unsigned int) * *number, hipMemcpyHostToDevice);
	free(tmpdata);
	return data;
}

void cudaFreepacket(char **data,int number) {
	char** tmpdata;
	tmpdata = (char **)malloc(sizeof(char *)*number);
	hipMemcpy(tmpdata, data, sizeof(char *)*number, hipMemcpyDeviceToHost);
	for (int i = 0; i < number; i++) {
		hipFree(tmpdata[i]);
	}
	hipFree(data);
	free(tmpdata);
}

void cudaFreepacket1(unsigned int *data) {
	hipFree(data);
}

void experiment_start() {
	uint32 len;
	string filename = "..\\..\\data\\srcdstsize.txt";
	int number=10000000;
	packetdata=readdata1(&number,filename,packetdata);
	int *cudanumber;
	hipMalloc((void**)&cudanumber, sizeof(int));
	hipMemcpy(cudanumber, &number, sizeof(int), hipMemcpyHostToDevice);
	initcurand << <1, 1 >> > ();
	hipError_t cudaStatus = hipDeviceSynchronize();
	for (int lll = 0; lll < number_of_test; lll++) {
		if(GSB_or_GVS[lll]==0)
			GSB = initSketchBloom(sketch_name[lll]);
		else
			if(GSB_or_GVS[lll] == 1)
				GVS =initVSketch(sketch_name[lll]);
		cudaStatus = hipDeviceSynchronize();
		hipEvent_t startEvent, stopEvent;
		hipEventCreate(&startEvent);
		hipEventCreate(&stopEvent);
		hipEventRecord(startEvent, 0);
		initsketch << <(number / 1024 + 1), 1024 >> > (GSB, GVS,  GSB_or_GVS[lll], size_or_spread[lll], number, packetdata);
		hipEventRecord(stopEvent, 0);
		cudaStatus = hipEventSynchronize(stopEvent);
		float time;
		hipEventElapsedTime(&time, startEvent, stopEvent);
		cout << "GVS or GSB="<<GSB_or_GVS[lll]<<" size_or_spread="<<size_or_spread[lll]<<" sketchname="<<sketch_name[lll] <<" time=" << number / time / 1000 << "million packets/ms" << endl;
		hipMemcpy(&number,cudanumber, sizeof(int), hipMemcpyDeviceToHost);
		cout << "number=" << number << endl;
		hipEventDestroy(startEvent);
		hipEventDestroy(stopEvent);
		//if you want output, please use these code
		/*/if (GSB_or_GVS[lll] == 0)
			getoutputGSB(GSB, sketch_name[lll], size_or_spread[lll]);
		else
			getoutputGVS(GVS,sketch_name[lll], size_or_spread[lll]);/*/
		if (GSB_or_GVS[lll] == 0)hipFree(GSB);
		else
			if (GSB_or_GVS[lll] == 1)
				hipFree(GVS);
	}
}

int main() {
	experiment_start();
	return 0;
}
